#include<stdio.h>

#include<hip/hip_runtime.h>


//vector addition kernel
__global__ void vadd(float *a, float *b, float *c, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < N){
    c[i] = a[i] + b[i];
  }
}

int main()
{

  int N = 50000; //vector elements
  hipError_t err = hipSuccess;
  
  //Allocate memory space using unified memory
  float *va, *vb, *vc;

  //pick one for error checking
  err = hipMallocManaged( &va, N*sizeof(float) );
  if(err != hipSuccess){
    printf("Failed to allocate using unified memory, error code %s. \n",
	   hipGetErrorString(err) );
    exit(EXIT_FAILURE);
  }
  
  err = hipMallocManaged( &vb, N*sizeof(float) );
 if(err != hipSuccess){
    printf("Failed to allocate using unified memory, error code %s. \n",
	   hipGetErrorString(err) );
    exit(EXIT_FAILURE);
  }
  
  hipMallocManaged( &vc, N*sizeof(float) );

  //Initiate value for each vector
  for(int i=0; i<N; ++i){
    va[i] = rand() / (float)RAND_MAX;
    vb[i] = rand() / (float)RAND_MAX;
    vc[i] = 0.0f;
  }

  //Launching kernel
  int threads = 256;
  int blocks = (N + threads - 1) / threads;
  printf("Blocks: %d, Threads: %d\n", blocks, threads);
  vadd<<<blocks, threads>>>(va, vb, vc, N);

  //Need to synchronize for data transfer
  hipDeviceSynchronize();
  
  err = hipGetLastError();
  if(err != hipSuccess){
    printf("Failed to launch kernel, error code %s\n", hipGetErrorString(err) );
  }
  
  //Checking answer
  float eps;
  for(int i=0; i<N; ++i){
    eps = va[i] + vb[i] - vc[i];
    if (eps > 1.0e-5){
      printf("Result verification failed at element %d \n", i);
      exit(EXIT_FAILURE);
    }
  }

  printf("Kernel and unified memory test success!\n");

  //Free memory; only required the use of cudaFree
  hipFree(va);
  hipFree(vb);
  hipFree(vc);
  
  
}

