#include<stdio.h>

#include<hip/hip_runtime.h>

/*
This is a simple vector addtion program.
C = A + B
*/

/*
CUDA Kernel Device Code starts with __global__ keyword
*/
__global__ void vadd( float *A, float *B, float *C, int N)
{

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N){
    C[i] = A[i] + B[i];
  }
  
}

/*Host function*/
int main()
{

  //call for error code to check CUDA calls
  hipError_t err = hipSuccess;

  //vector length
  int N = 50000;
  size_t size = N * sizeof(float);
  printf("Vector addition of %d elements.\n", N);

  //Allocate memory for host
  float *h_A = (float *)malloc(size);
  float *h_B = (float *)malloc(size);
  float *h_C = (float *)malloc(size);
  
  //Error check

  //Initialized value for host vectors
  for(int i=0; i< N; ++i){
    h_A[i] = rand() / (float)RAND_MAX;
    h_B[i] = rand() / (float)RAND_MAX;
    h_C[i] = 0.0f; //good practice in numerical method
  }
  
  //allocate device vectors
  float *d_A = NULL;
  err = hipMalloc( (void **)&d_A, size ); //err to check for error

  float *d_B = NULL;
  err = hipMalloc( (void **)&d_B, size ); //err to check for error

  float *d_C = NULL;
  err = hipMalloc( (void **)&d_C, size ); //err to check for error

  //No error check; could be fatal but let skip it for a while

  //Copy host vectors h_A, h_B to device vectors d_A, d_B
  printf("Copy input data from host memory to the CUDA device.\n");

  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  //Launch the vector addition kernel
  int threadsperblock = 256;
  int blockspergrid = (N + threadsperblock-1) / threadsperblock;
  printf("Launching CUDA kernel with %d blocks and %d threads.\n", blockspergrid,
	 threadsperblock);

  vadd<<<blockspergrid, threadsperblock >>>(d_A, d_B, d_C, N); //the kernel call

  err = hipGetLastError();
  //minimal checking; at least need to know kernel success.
  if(err != hipSuccess)
  {
    printf("Failed to launch vadd kernel, error code %s.\n",
	   hipGetErrorString(err) );
    exit(EXIT_FAILURE);
  }

  //Copy device result vector to host result vector in host memory
  printf("Copying output data from CUDA device to host memory.\n");
  err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  if(err != hipSuccess)
    {
      printf("Failed to copy data from device, error %s.\n", hipGetErrorString(err) );
      exit(EXIT_FAILURE);
    }

  //verify result
  for(int i=0; i<N; ++i){

    if( fabs(h_A[i] + h_B[i] - h_C[i] ) > 1.0e-5 ){
      printf("Result verification failed at element %d \n", i);
      exit(EXIT_FAILURE);
    }
  }

  printf("Kernel success!\n");

  //Free device global memory
  err = hipFree(d_A);
  err = hipFree(d_B);
  err = hipFree(d_C);

  //Free host memory
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
        
}
